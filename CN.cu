
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <string>
#include <fstream>
#include <iostream>
#include <algorithm>
#include <sstream>
#include <assert.h>
using namespace std;

// includes, project
//#include <cutil.h>

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int, unsigned int);
void NeuralNetwork();

unsigned g_verbose;
unsigned NUM;

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
	int i, commandline_error;
	commandline_error = 0;
	g_verbose = 0;
	if (argc >= 2) {
		NUM = atoi(argv[1]);
		for (i=2; i < argc;i++) {
			if (argv[i][0] == '-') {
				switch (argv[i][1]) {
				case 'v': g_verbose = 1;
					break;
				default: commandline_error=1;
				}
			}
			else commandline_error=1;
		}
	} else commandline_error=1;

	if (commandline_error || !NUM) {
		printf("Usage: ./NN <NUM> [-v]\n");
		printf("where NUM is the number of images to process in parallel (up to 10000 for the t10k-images-idx3-ubyte database file) and -v is used to display approximately what each image looks like.\n");
		return 1;
	}


	NeuralNetwork();
    //CUT_EXIT(argc, argv);
}

void InitHostMem(double *Layer1_Weights_CPU, double *Layer2_Weights_CPU)
{
	// initial layer 1 weight
	FILE * pFile1 = fopen ("data/conv1.txt","rb");
	if (pFile1 != NULL)
	{
		printf("File Opened\n");
		char s[300000] = "";
		fread(s,sizeof(s),1,pFile1);
		printf("Done2\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			double temp_num = atof(temp_string);
			Layer1_Weights_CPU[i] = temp_num;
			i++;
			index++;
			if(i==2400)
			{
				printf("Breaking\n");
				break;
			}
			temp_string = strtok(NULL, delim);
		}
		fclose (pFile1);
	}	
	
	if (!pFile1)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
	//Layer 2 Weights
	FILE * pFile2 = fopen ("data/conv2.txt","rb");
	if (pFile2 != NULL)
	{
		printf("File 2 Opened\n");
		char s[3000000] = "";
		fread(s,sizeof(s),1,pFile2);
		//printf("%s\n",s);
		printf("Done\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			double temp_num = atof(temp_string);
			//printf("%.8f %d\n",temp_num,i);
			Layer2_Weights_CPU[i] = temp_num;
			i++;
			index++;
			if(i==25600)
			{
				printf("Breaking\n");
				break;
			}
			temp_string = strtok(NULL, delim);
		}
		//printf("Last Value: %.8f\n",Layer2_Weights_CPU[25599]);
		fclose (pFile2);
	}	
	
	if (!pFile2)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
}

void LoadInput(int *Data_Layer_CPU)
{
	FILE * pFile1 = fopen ("data/speed-limit-35.txt","rb");
	if (pFile1 != NULL)
	{
		printf("File Opened\n");
		char s[300000] = "";
		fread(s,sizeof(s),1,pFile1);
		//printf("%s", s);
		printf("Done2\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		//int address = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			int temp_num = atof(temp_string);	
			Data_Layer_CPU[i] = temp_num;
			i++;
			index++;
			if(i==(32*32*3))
			{
				printf("Breaking input\n");
				break;
			}
			temp_string = strtok(NULL, delim);
			//if(temp_string != NULL)
			//	address = strlen(temp_string);
		}
		//printf("%d", Data_Layer_CPU[(32*32*3)-1]);
		fclose (pFile1);
		//printf("Last image value: %d", );
	}	
	if (!pFile1)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
}

void ConvertInput(int *Data_Layer_CPU_R, int *Data_Layer_CPU_G, int *Data_Layer_CPU_B, int *Data_Layer_CPU)
{
	for(int i=0; i<32*32*3; i+=3)
	{
		Data_Layer_CPU_R[i/3] = Data_Layer_CPU[i];
		Data_Layer_CPU_G[i/3] = Data_Layer_CPU[i+1];
		Data_Layer_CPU_B[i/3] = Data_Layer_CPU[i+2];
	}
}
void ExecuteFirstLayer(double *Layer1_Weights_CPU, int *Data_Layer_CPU_R, int *Data_Layer_CPU_G, int *Data_Layer_CPU_B, double ***Layer1_Features)
{
	for(int f=0; f<32; f++)
	{
		double maskR[25], maskG[25], maskB[25];
		int imageR[25], imageG[25], imageB[25];
		for(int i=0; i<25; i++)
		{
			maskR[i] = Layer1_Weights_CPU[i+f*75];
			maskG[i] = Layer1_Weights_CPU[i+25+f*75];
			maskB[i] = Layer1_Weights_CPU[i+50+f*75];
		}
		for(int x=0; x<32; x++)
		{
			for(int y=0; y<32; y++)
			{
				for(int i = x-2; i<=x+2; i++)
				{
    					for(int j=y-2; j<=y+2; j++)
    					{
						int x_index = i-x+2;
						int y_index = j-y+2;
         					if(i<0 || j<0)
						{
             				 		imageR[(y_index)+(x_index)*5] = 0;
							imageG[(y_index)+(x_index)*5] = 0;
							imageB[(y_index)+(x_index)*5] = 0;
						}
         					else if(j>31 || i>31)
						{
              						imageR[(y_index)+(x_index)*5] = 0;
							imageG[(y_index)+(x_index)*5] = 0;
							imageB[(y_index)+(x_index)*5] = 0;
						}
         					else
						{
               						imageR[(y_index)+(x_index)*5] = Data_Layer_CPU_R[(y_index-2) + x*32 + y + (x_index-2)*32];
               						imageG[(y_index)+(x_index)*5] = Data_Layer_CPU_G[(y_index-2) + x*32 + y + (x_index-2)*32];
               						imageB[(y_index)+(x_index)*5] = Data_Layer_CPU_B[(y_index-2) + x*32 + y + (x_index-2)*32];			
						}
					}
				}
				double result = 0;
				for(int i=0; i<25; i++)
				{
					 result+= imageR[i]*maskR[i] +imageG[i]*maskG[i] + imageB[i]*maskB[i]; 
				}   
				Layer1_Features[f][x][y] = result;
				//printf("%f ", result);
			}
		}
	}
	printf("\n");
	for(int x=0; x<32; x++)
	{
		for(int y=0; y<32; y++)
		{
			//printf("%.8f  %d\n",Layer1_Features[1][x][y], x*32+y);
		}
		//printf("\n");
	}
}

void ExecuteSecondLayer(double *Layer2_Weights_CPU, double ***Layer2_Features, double ***Layer2_pool_GPU)
{
	printf("Second Layer Executions:\n");
	for(int f=0; f<32; f++)
	{
		double mask[32][25];
		double input[32][25];
		double Features[32][16][16];
		for(int n=0; n<32; n++)
		{
			for(int i=0; i<25; i++)
			{
				mask[n][i] = Layer2_Weights_CPU[i+f*25*32+n*25];
				//printf("%.8f ", mask[n][i]);
			}
			//printf("\n");
		}
		//printf("Weights Load Complete\n");
		for(int n=0; n<32; n++)
		{
			for(int x=0; x<16; x++)
			{
				for(int y=0; y<16; y++)
				{
					for(int i = x-2; i<=x+2; i++)
					{
    						for(int j=y-2; j<=y+2; j++)
    						{
							int x_index = i-x+2;
							int y_index = j-y+2;
         						if(i<0 || j<0)
							{
             					 		input[n][(y_index)+(x_index)*5] = 0;
							}
         						else if(j>15 || i>15)
							{
              							input[n][(y_index)+(x_index)*5] = 0;
							}
         						else
							{
               							input[n][(y_index)+(x_index)*5] = Layer2_pool_GPU[n][x_index+x-2][y_index+y-2];			
							}
						}
					}
					double result = 0;
					for(int i=0; i<25; i++)
					{
						result+= input[n][i]*mask[n][i]; 
						//printf("%.8f ",input[n][i]);
					}   
					Features[n][x][y] = result;
					//printf("%f [%d][%d][%d]\n", result,n,x,y);
				}
			}
		}
		for(int n=0; n<32; n++)
		{
			for(int x=0; x<16; x++)
			{
				for(int y=0; y<16; y++)
				{
					Layer2_Features[f][x][y]+= Features[n][x][y];
				}
			}
		}
	}
	for(int f=0; f<32; f++)
	{
		for(int x=0; x<16; x++)
		{
			for(int y=0; y<16; y++)
			{
				if(Layer2_Features[f][x][y] < 0)
					Layer2_Features[f][x][y] = 0;
			}	
		}
		//printf("\n");
	}
	//printf("First Value: %.8f\n",Layer2_Features[31][15][11]);
}

void pooling1(double ***Layer2_Neurons_GPU,double ***Layer2_pool_GPU,int out,int out_fr,int out_fc,int kernel,int stride_width,int in_fr,int in_fc)
{
    printf("pooling Activation layer \n");
    double max = 0.0;
    {
        for(int output =0;output < out ;output++)
        {
            for(int row =1; row <= 31 ;row+=2)
            { 
                for(int col =1; col <= 31 ;col+=2)
                {
                    for(int i = row-1; i <= row+1; i++)
                    {   
			if(i>31) break;        
                        for(int j = col-1; j <= col+1; j++)
                        {
			    if(j>31) break;
                            if(max < ((Layer2_Neurons_GPU[output][i][j])))
                                max =   ((Layer2_Neurons_GPU[output][i][j])) ;

                        }
                    }
		    if(max<0)
			max = 0;
                    Layer2_pool_GPU[output][(row-1)/2][(col-1)/2] = max;
                    //printf("%f %d \n",max, (((row-1)*8)+((col-1)/2) + output*16*16));     
                    max = 0.0;   
                }
            }
	//printf("\n");
        }
    }
}

void pooling2(double ***Layer2_Neurons_GPU,double ***Layer2_pool_GPU,int out,int out_fr,int out_fc,int kernel,int stride_width,int in_fr,int in_fc)
{
    printf("pooling 2 layer \n");
    double avg = 0.0;
    int count = 0;
    {
        for(int output =0;output < out ;output++)
        {
            for(int row =1; row <= 16 ;row+=2)
            { 
                for(int col =1; col <= 16 ;col+=2)
                {
                    for(int i = row-1; i <= row+1; i++)
                    {   
			if(i>15) break;        
                        for(int j = col-1; j <= col+1; j++)
                        {
			    if(j>15) break;
                            avg+= ((Layer2_Neurons_GPU[output][i][j]));
			    count++;

                        }
                    }
                    Layer2_pool_GPU[output][(row-1)/2][(col-1)/2] = avg/count;
                    //printf("%f %d \n",max, (((row-1)*8)+((col-1)/2) + output*16*16));     
                    avg = 0.0;   
		    count=0;
                }
            }
	//printf("\n");
        }
    }
    for(int i=0; i<8; i++)
    	printf("%.8f ",Layer2_pool_GPU[31][7][i]);
    printf("\n");
}

void NeuralNetwork()
{
	double *Layer1_Weights_CPU = (double*) malloc (3*32*32* NUM * sizeof(double));
	double *Layer2_Weights_CPU = (double*) malloc (5*5*32*32* NUM * sizeof(double));
	int *Data_Layer_CPU_R = (int*) malloc (32*32*NUM*sizeof(int));
	int *Data_Layer_CPU_G = (int*) malloc (32*32*NUM*sizeof(int));
	int *Data_Layer_CPU_B = (int*) malloc (32*32*NUM*sizeof(int));
	int *Data_Layer_CPU = (int*) malloc (3*32*32*NUM*sizeof(int));
	InitHostMem(Layer1_Weights_CPU, Layer2_Weights_CPU);
	LoadInput(Data_Layer_CPU);
	ConvertInput(Data_Layer_CPU_R, Data_Layer_CPU_G, Data_Layer_CPU_B, Data_Layer_CPU);
	double ***Layer1_Features;
	Layer1_Features = (double***)malloc(32*sizeof(double **));
	assert(Layer1_Features!= NULL);
	for(int i=0; i<32; i++)
	{
		Layer1_Features[i] = (double**)malloc(32*sizeof(double *));
		assert(Layer1_Features[i] != NULL);
		for(int j=0; j<32; j++)
		{
			Layer1_Features[i][j] = (double*)malloc(32*sizeof(double));
		}
	}
	ExecuteFirstLayer(Layer1_Weights_CPU, Data_Layer_CPU_R, Data_Layer_CPU_G, Data_Layer_CPU_B, Layer1_Features);
	double ***Pool_Layer_Features;
	Pool_Layer_Features = (double***)malloc(32*sizeof(double **));
	assert(Pool_Layer_Features!= NULL);
	for(int i=0; i<32; i++)
	{
		Pool_Layer_Features[i] = (double**)malloc(16*sizeof(double *));
		assert(Pool_Layer_Features[i] != NULL);
		for(int j=0; j<16; j++)
		{
			Pool_Layer_Features[i][j] = (double*)malloc(16*sizeof(double));
		}
	}
	pooling1(Layer1_Features, Pool_Layer_Features, 32, 16, 16, 5, 2, 32, 32);
	double ***Layer2_Features;
	Layer2_Features = (double***)malloc(32*sizeof(double **));
	assert(Layer2_Features!= NULL);
	for(int i=0; i<32; i++)
	{
		Layer2_Features[i] = (double**)malloc(16*sizeof(double *));
		assert(Layer2_Features[i] != NULL);
		for(int j=0; j<16; j++)
		{
			Layer2_Features[i][j] = (double*)malloc(16*sizeof(double));
		}
	}
	ExecuteSecondLayer(Layer2_Weights_CPU, Layer2_Features, Pool_Layer_Features);
	double ***Pool2_Layer_Features;
	Pool2_Layer_Features = (double***)malloc(32*sizeof(double **));	
	assert(Pool2_Layer_Features!= NULL);
	for(int i=0; i<32; i++)
	{
		Pool2_Layer_Features[i] = (double**)malloc(8*sizeof(double *));
		assert(Pool2_Layer_Features[i] != NULL);
		for(int j=0; j<8; j++)
		{
			Pool2_Layer_Features[i][j] = (double*)malloc(8*sizeof(double));
		}
	}
	pooling2(Layer2_Features, Pool2_Layer_Features, 32, 8, 8, 5, 2, 16, 16);
}


