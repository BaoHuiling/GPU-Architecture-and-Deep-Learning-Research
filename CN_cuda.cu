#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <string>
#include <fstream>
#include <iostream>
#include <algorithm>
#include <sstream>
#include <assert.h>
using namespace std;

// includes, project
//#include <cutil.h>

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int, unsigned int);
void NeuralNetwork();

unsigned g_verbose;
unsigned NUM;

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
	int i, commandline_error;
	commandline_error = 0;
	g_verbose = 0;
	if (argc >= 2) {
		NUM = atoi(argv[1]);
		for (i=2; i < argc;i++) {
			if (argv[i][0] == '-') {
				switch (argv[i][1]) {
				case 'v': g_verbose = 1;
					break;
				default: commandline_error=1;
				}
			}
			else commandline_error=1;
		}
	} else commandline_error=1;

	if (commandline_error || !NUM) {
		printf("Usage: ./NN <NUM> [-v]\n");
		printf("where NUM is the number of images to process in parallel (up to 10000 for the t10k-images-idx3-ubyte database file) and -v is used to display approximately what each image looks like.\n");
		return 1;
	}


	NeuralNetwork();
    //CUT_EXIT(argc, argv);
}

void InitHostMem(double *Layer1_Weights_CPU, double *Layer2_Weights_CPU, double *Layer3_Weights_CPU, double *Layer4_Weights_CPU, double *Layer5_Weights_CPU)
{
	// initial layer 1 weight
	FILE * pFile1 = fopen ("data/conv1.txt","rb");
	if (pFile1 != NULL)
	{
		printf("File Opened\n");
		char s[300000] = "";
		fread(s,sizeof(s),1,pFile1);
		printf("Done2\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			double temp_num = atof(temp_string);
			Layer1_Weights_CPU[i] = temp_num;
			i++;
			index++;
			if(i==2400)
			{
				printf("Breaking\n");
				break;
			}
			temp_string = strtok(NULL, delim);
		}
		fclose (pFile1);
	}	
	
	if (!pFile1)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
	//Layer 2 Weights
	FILE * pFile2 = fopen ("data/conv2.txt","rb");
	if (pFile2 != NULL)
	{
		printf("File 2 Opened\n");
		char s[3000000] = "";
		fread(s,sizeof(s),1,pFile2);
		//printf("%s\n",s);
		printf("Done\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			double temp_num = atof(temp_string);
			//printf("%.8f %d\n",temp_num,i);
			Layer2_Weights_CPU[i] = temp_num;
			i++;
			index++;
			if(i==25600)
			{
				printf("Breaking\n");
				break;
			}
			temp_string = strtok(NULL, delim);
		}
		//printf("Last Value: %.8f\n",Layer2_Weights_CPU[25599]);
		fclose (pFile2);
	}	
	
	if (!pFile2)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
	//Layer 3 Weights
	FILE * pFile3 = fopen ("data/conv3.txt","rb");
	if (pFile3 != NULL)
	{
		printf("File 3 Opened\n");
		char s[6000000] = "";
		fread(s,sizeof(s),1,pFile2);
		//printf("%s\n",s);
		printf("Done\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			double temp_num = atof(temp_string);
			//printf("%.8f %d\n",temp_num,i);
			Layer3_Weights_CPU[i] = temp_num;
			i++;
			index++;
			if(i==51200)
			{
				printf("Breaking\n");
				break;
			}
			temp_string = strtok(NULL, delim);
		}
		//printf("Last Value: %.8f\n",Layer3_Weights_CPU[51100]);
		fclose (pFile3);
	}	
	
	if (!pFile3)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
	//Layer 4 Weights
	FILE * pFile4 = fopen ("data/ip1.txt","rb");
	if (pFile4 != NULL)
	{
		printf("File 4 Opened\n");
		char s[8000000] = "";
		fread(s,sizeof(s),1,pFile2);
		//printf("%s\n",s);
		printf("Done\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			double temp_num = atof(temp_string);
			//printf("%.8f %d\n",temp_num,i);
			Layer4_Weights_CPU[i] = temp_num;
			i++;
			index++;
			if(i==65536)
			{
				printf("Breaking\n");
				break;
			}
			temp_string = strtok(NULL, delim);
		}
		//printf("First Value: %.8f\n",Layer4_Weights_CPU[0]);
		fclose (pFile4);
	}	
	
	if (!pFile4)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
	//Layer 5 Weights
	FILE * pFile5 = fopen ("data/ip2.txt","rb");
	if (pFile5 != NULL)
	{
		printf("File 5 Opened\n");
		char s[80000] = "";
		fread(s,sizeof(s),1,pFile2);
		//printf("%s\n",s);
		printf("Done\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			double temp_num = atof(temp_string);
			//printf("%.8f %d\n",temp_num,i);
			Layer5_Weights_CPU[i] = temp_num;
			i++;
			index++;
			if(i==576)
			{
				printf("Breaking\n");
				break;
			}
			temp_string = strtok(NULL, delim);
		}
		//printf("Last Value: %.8f\n",Layer5_Weights_CPU[575]);
		fclose (pFile5);
	}	
	
	if (!pFile5)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
}

void LoadInput(int *Data_Layer_CPU)
{
	FILE * pFile1 = fopen ("data/speed-limit-35.txt","rb");
	if (pFile1 != NULL)
	{
		printf("File Opened\n");
		char s[300000] = "";
		fread(s,sizeof(s),1,pFile1);
		//printf("%s", s);
		printf("Done2\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		//int address = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			int temp_num = atof(temp_string);	
			Data_Layer_CPU[i] = temp_num;
			i++;
			index++;
			if(i==(32*32*3))
			{
				printf("Breaking input\n");
				break;
			}
			temp_string = strtok(NULL, delim);
			//if(temp_string != NULL)
			//	address = strlen(temp_string);
		}
		//printf("%d", Data_Layer_CPU[(32*32*3)-1]);
		fclose (pFile1);
		//printf("Last image value: %d", );
	}	
	if (!pFile1)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
}

void ConvertInput(int *Data_Layer_CPU_R, int *Data_Layer_CPU_G, int *Data_Layer_CPU_B, int *Data_Layer_CPU)
{
	for(int i=0; i<32*32*3; i+=3)
	{
		Data_Layer_CPU_R[i/3] = Data_Layer_CPU[i];
		Data_Layer_CPU_G[i/3] = Data_Layer_CPU[i+1];
		Data_Layer_CPU_B[i/3] = Data_Layer_CPU[i+2];
	}
}
__global__ void ExecuteFirstLayer(double *Layer1_Weights_CPU, int *Data_Layer_CPU_R, int *Data_Layer_CPU_G, int *Data_Layer_CPU_B, double *Layer1_Features)
{
	//printf("First Layer Execution\n");
	int tid = threadIdx.x + threadIdx.y*32;
	int x = threadIdx.x;
	int y = threadIdx.y;
	for(int f=0; f<32; f++)
	{
				double result = 0;
				for(int i = x-2; i<=x+2; i++)
				{
    					for(int j=y-2; j<=y+2; j++)
    					{
						int x_index = i-x+2;
						int y_index = j-y+2;
						int m = (y_index)+(x_index)*5;
         					if(i<0 || j<0)
						{
							result+= 0;						
						}
         					else if(j>31 || i>31)
						{
							result+= 0;
						}
         					else
						{
							result += Data_Layer_CPU_R[(y_index-2) + x*32 + y + (x_index-2)*32]*Layer1_Weights_CPU[m+f*75] + Data_Layer_CPU_G[(y_index-2) + x*32 + y + (x_index-2)*32]*Layer1_Weights_CPU[m+25+f*75] + Data_Layer_CPU_B[(y_index-2) + x*32 + y + (x_index-2)*32]*Layer1_Weights_CPU[m+50+f*75];			
						}
					}
				} 
				Layer1_Features[f*32*32+x*32+y] = result;
	}
}

__global__ void ExecuteSecondLayer(double *Layer2_Weights_CPU, double *Layer2_Features, double *Layer2_pool_GPU)
{
	//printf("Second Layer Execution\n");
	double Features = 0;
	int x = threadIdx.x;
	int y = threadIdx.y;
	for(int f=0; f<32; f++)
	{
		Features = 0;
		//double mask[32][25];
		//double input[32][25];
		//double Features[32][16][16];
		/*for(int n=0; n<32; n++)
		{
			for(int i=0; i<25; i++)
			{
				mask[n][i] = Layer2_Weights_CPU[i+f*25*32+n*25];
				//printf("%.8f ", mask[n][i]);
			}
			//printf("\n");
		}*/
		//printf("Weights Load Complete\n");
		for(int n=0; n<32; n++)
		{
			if(x<16)//for(int x=0; x<16; x++)
			{
				if(y<16)//for(int y=0; y<16; y++)
				{
					double result = 0;
					for(int i = x-2; i<=x+2; i++)
					{
    						for(int j=y-2; j<=y+2; j++)
    						{
							int x_index = i-x+2;
							int y_index = j-y+2;
							int m = (y_index)+(x_index)*5;
         						if(i<0 || j<0)
							{
             					 		//input[n][(y_index)+(x_index)*5] = 0;
								result+=0;
							}
         						else if(j>15 || i>15)
							{
              							//input[n][(y_index)+(x_index)*5] = 0;
								result+=0;	
							}
         						else
							{
               							result+= Layer2_pool_GPU[n*16*16 + (x_index+x-2)*16 + (y_index+y-2)]*Layer2_Weights_CPU[m+f*25*32+n*25];			
							}
						}
					}
					/*for(int i=0; i<25; i++)
					{
						result+= input[n][i]*mask[n][i]; 
						//printf("%.8f ",input[n][i]);
					} */  
					Features += result;
					//printf("%f [%d][%d][%d]\n", result,n,x,y);
				}
			}
		}
		if(Features<0)
			Features = 0;
		Layer2_Features[f*16*16 + x*16 + y] = Features;
		/*if((x==0) && (y==0))
			printf("%.8f\n",Features);*/
	}
}

__global__ void ExecuteThirdLayer(double *Layer3_Weights_CPU, double *Layer3_Features, double *Layer3_pool_GPU)
{
	//printf("Third Layer Execution\n");
	double Features = 0;
	int x = threadIdx.x;
	int y = threadIdx.y;
	for(int f=0; f<64; f++)
	{
		Features = 0;
		/*double mask[32][25];
		double input[32][25];
		double Features[64][8][8];
		for(int n=0; n<32; n++)
		{
			for(int i=0; i<25; i++)
			{
				mask[n][i] = Layer3_Weights_CPU[i+f*25*32+n*25];
				//printf("%.8f ", mask[n][i]);
			}
			//printf("\n");
		}*/
		//printf("Weights Load Complete\n");
		for(int n=0; n<32; n++)
		{
			if(x<8)//for(int x=0; x<8; x++)
			{
				if(y<8)//for(int y=0; y<8; y++)
				{
					double result = 0;
					for(int i = x-2; i<=x+2; i++)
					{
    						for(int j=y-2; j<=y+2; j++)
    						{
							int x_index = i-x+2;
							int y_index = j-y+2;
							int m = (y_index)+(x_index)*5;
         						if(i<0 || j<0)
							{
             					 		//input[n][(y_index)+(x_index)*5] = 0;
								result+=0;
							}
         						else if(j>7 || i>7)
							{
              							//input[n][(y_index)+(x_index)*5] = 0;
								result+=0;
							}
         						else
							{
               							result+= Layer3_pool_GPU[n*8*8 + (x_index+x-2)*8 + (y_index+y-2)]*Layer3_Weights_CPU[m+f*25*32+n*25];			
							}
						}
					}
					//double result = 0;
					/*for(int i=0; i<25; i++)
					{
						result+= input[n][i]*mask[n][i]; 
						//printf("%.8f ",input[n][i]);
					} */  
					Features += result;
					//printf("%f [%d][%d][%d]\n", result,n,x,y);
				}
			}
		}
		if(Features<0)
			Features = 0;
		Layer3_Features[f*8*8 + x*8 + y] = Features;
		//if((x==0) && (y==0))
			//printf("%.8f\n",Features);
		/*for(int n=0; n<32; n++)
		{
			for(int x=0; x<8; x++)
			{
				for(int y=0; y<8; y++)
				{
					Layer3_Features[f][x][y]+= Features[n][x][y];
				}
			}
		}*/
	}
	/*for(int f=0; f<64; f++)
	{
		for(int x=0; x<8; x++)
		{
			for(int y=0; y<8; y++)
			{
				if(Layer3_Features[f][x][y] < 0)
					Layer3_Features[f][x][y] = 0;
			}	
		}
		//printf("\n");
	}*/
	//printf("First Value: %.8f\n",Layer3_Features[63][4][0]);
}

__global__ void ExecuteFourthLayer(double *Layer4_Weights_CPU, double *Layer4_Features, double *Pool3_Layer_Features)
{
	//printf("Fourth Layer Execution\n");
	int n = threadIdx.x;
	//for(int n=0;n<64; n++)
	{
		double result = 0;
		for(int f=0; f<64; f++)
		{
			for(int x=0; x<4; x++)
			{
				for(int y=0; y<4; y++)
				{
					result+= Pool3_Layer_Features[f*4*4 +x*4 + y] * Layer4_Weights_CPU[y+(x*4)+(f*4*4)+(n*4*4*64)];
				}
			}
		}
		Layer4_Features[n] = result;
		//printf("%.8f ",result);
		//result = 0;
	}
	//printf("\n");
	//if(n==0)
		//printf("%.8f", Layer4_Features[n]);
}

__global__ void ExecuteFifthLayer(double *Layer5_Weights_CPU, double *Layer5_Features, double *Layer4_Features)
{
	//printf("Fifth Layer Execution\n");
	int n = threadIdx.x;
	if(n<9)//for(int n=0;n<9; n++)
	{
		double result = 0;
		for(int f=0; f<64; f++)
		{
			result+= Layer4_Features[f] * Layer5_Weights_CPU[f+n*64];
		}
		Layer5_Features[n] = result;
		printf("%.8f ",result);
		result = 0;
	}
	//printf("\n");
	//if(n==0)
		//printf("%.8f", Layer5_Features[n]);
}

__global__ void pooling1(double *Layer2_Neurons_GPU,double *Layer2_pool_GPU,int out,int out_fr,int out_fc,int kernel,int stride_width,int in_fr,int in_fc)
{
    //printf("pooling Activation layer \n");
    int row = threadIdx.x;
    int col = threadIdx.y;
    double max = 0.0;
    {
        for(int output =0;output < out ;output++)
        {
            if(row%2 != 0)//for(int row =1; row <= 31 ;row+=2)
            { 
                if(col%2 != 0)//for(int col =1; col <= 31 ;col+=2)
                {
                    for(int i = row-1; i <= row+1; i++)
                    {   
			if(i>31) break;        
                        for(int j = col-1; j <= col+1; j++)
                        {
			    if(j>31) break;
                            if(max < ((Layer2_Neurons_GPU[output*32*32+i*32+j])))
                                max =   ((Layer2_Neurons_GPU[output*32*32+i*32+j])) ;

                        }
                    }
		    if(max<0)
			max = 0;
                    Layer2_pool_GPU[output*16*16+(row-1)*8+(col-1)/2] = max;
                    //printf("%f %d \n",max, (((row-1)*8)+((col-1)/2) + output*16*16));
		        /*if(row == 1 && col == 1)
    			{
			     printf("%.8f\n",max);
    			}  */   
                    max = 0.0;   
                }
            }
	//printf("\n");
        }
    }
    /*if(row == 0 && col == 0)
    {
	printf("%.8f\n",max);
    }*/
}

__global__ void pooling2(double *Layer2_Neurons_GPU,double *Layer2_pool_GPU,int out,int out_fr,int out_fc,int kernel,int stride_width,int in_fr,int in_fc)
{
    //printf("pooling 2 layer \n");
    double avg = 0.0;
    int count = 0;
    int row = threadIdx.x;
    int col = threadIdx.y;
    {
        for(int output =0;output < out ;output++)
        {
            if((row%2 != 0) && (row<16))//for(int row =1; row <= 16 ;row+=2)
            { 
                if((col%2 != 0) && (col<16))//for(int col =1; col <= 16 ;col+=2)
                {
                    for(int i = row-1; i <= row+1; i++)
                    {   
			if(i>15) break;        
                        for(int j = col-1; j <= col+1; j++)
                        {
			    if(j>15) break;
                            avg+= Layer2_Neurons_GPU[output*16*16 + i*16 + j];
			    count = count + 1;

                        }
                    }
                    Layer2_pool_GPU[output*8*8+(row-1)*4+(col-1)/2] = avg/count;
                    //printf("%f %d \n",max, (((row-1)*8)+((col-1)/2) + output*16*16));     
                    avg = 0.0;   
		    count=0;
                }
            }
	//printf("\n");
        }
    }
    //for(int i=0; i<8; i++)
    	//printf("%.8f ",Layer2_pool_GPU[31][7][i]);
    //printf("\n");
}

__global__ void pooling3(double *Layer3_Neurons_GPU,double *Layer3_pool_GPU,int out,int out_fr,int out_fc,int kernel,int stride_width,int in_fr,int in_fc)
{
    //printf("pooling 3 layer \n");
    double avg = 0.0;
    int count = 0;
    int row = threadIdx.x;
    int col = threadIdx.y;
    {
        for(int output =0;output < out ;output++)
        {
            if((row%2 != 0) && (row<8))//for(int row =1; row <= 8 ;row+=2)
            { 
                if((col%2 != 0) && (col<8))//for(int col =1; col <= 8 ;col+=2)
                {
                    for(int i = row-1; i <= row+1; i++)
                    {   
			if(i>7) break;        
                        for(int j = col-1; j <= col+1; j++)
                        {
			    if(j>7) break;
                            avg+= ((Layer3_Neurons_GPU[output*8*8 + i*8 + j]));
			    count++;

                        }
                    }
                    Layer3_pool_GPU[output*4*4+(row-1)*2+(col-1)/2] = avg/count;
                    //printf("%f %d \n",max, (((row-1)*8)+((col-1)/2) + output*16*16));     
                    avg = 0.0;   
		    count=0;
                }
            }
	//printf("\n");
        }
    }
    /*for(int i=0; i<4; i++)
    	printf("%.8f ",Layer3_pool_GPU[63][3][i]);
    printf("\n");*/
    /*if(row == 0 && col == 0)
    {
	printf("%.8f\n",Layer3_pool_GPU[0]);
    }*/
}

void NeuralNetwork()
{
	hipError_t err;
	int deviceCount;                                                         
	hipGetDeviceCount(&deviceCount);                
	if (deviceCount == 0) {                                                  
		fprintf(stderr, "There is no device.\n");                            
		exit(EXIT_FAILURE);                                                  
	}                                                                        
	int dev;                                                                 
	for (dev = 0; dev < deviceCount; ++dev) {                                
		hipDeviceProp_t deviceProp;                                           
		hipGetDeviceProperties(&deviceProp, dev);   
		if (deviceProp.major >= 1)                                           
			break;                                                           
	}                                                                        
	if (dev == deviceCount) {                                                
		fprintf(stderr, "There is no device supporting CUDA.\n");            
		exit(EXIT_FAILURE);                                                  
	}                                                                        
	else                                                                     
		hipSetDevice(dev);
	//printf("Started");
	double *Layer1_Weights_CPU = (double*) malloc (3*32*32* NUM * sizeof(double));
	double *Layer2_Weights_CPU = (double*) malloc (5*5*32*32* NUM * sizeof(double));
	double *Layer3_Weights_CPU = (double*) malloc (5*5*32*64* NUM * sizeof(double));
	double *Layer4_Weights_CPU = (double*) malloc (64*4*4*64* NUM * sizeof(double));
	double *Layer5_Weights_CPU = (double*) malloc (64*9* NUM * sizeof(double));
	int *Data_Layer_CPU_R = (int*) malloc (32*32*NUM*sizeof(int));
	int *Data_Layer_CPU_G = (int*) malloc (32*32*NUM*sizeof(int));
	int *Data_Layer_CPU_B = (int*) malloc (32*32*NUM*sizeof(int));
	int *Data_Layer_GPU_R = (int*) malloc (32*32*NUM*sizeof(int));
	int *Data_Layer_GPU_G = (int*) malloc (32*32*NUM*sizeof(int));
	int *Data_Layer_GPU_B = (int*) malloc (32*32*NUM*sizeof(int));
	int *Data_Layer_CPU = (int*) malloc (3*32*32*NUM*sizeof(int));
	InitHostMem(Layer1_Weights_CPU, Layer2_Weights_CPU, Layer3_Weights_CPU, Layer4_Weights_CPU, Layer5_Weights_CPU);
	LoadInput(Data_Layer_CPU);
	ConvertInput(Data_Layer_CPU_R, Data_Layer_CPU_G, Data_Layer_CPU_B, Data_Layer_CPU);
	double *Layer1_Features;
	double *Layer1_Weights_GPU;
	err = hipMalloc((void**) &Layer1_Features, 32*32*32* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	err = hipMalloc((void**) &Layer1_Weights_GPU, 2400* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	err = hipMalloc((void**) &Data_Layer_GPU_R, 32*32* NUM * sizeof(int));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	err = hipMalloc((void**) &Data_Layer_GPU_G, 32*32* NUM * sizeof(int));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	err = hipMalloc((void**) &Data_Layer_GPU_B, 32*32* NUM * sizeof(int));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	printf("Malloc completed\n");
	hipMemcpy(Layer1_Weights_GPU,Layer1_Weights_CPU, sizeof(double)*2400*NUM, hipMemcpyHostToDevice);
	hipMemcpy(Data_Layer_GPU_R,Data_Layer_CPU_R, 32*32* NUM * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(Data_Layer_GPU_G,Data_Layer_CPU_G, 32*32* NUM * sizeof(int), hipMemcpyHostToDevice);
	err = hipMemcpy(Data_Layer_GPU_B,Data_Layer_CPU_B, 32*32* NUM * sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	printf("Memcpy completed\n");
	dim3 n_threads(32,32,1);
	dim3 n_blocks(1,1,1); 
	hipDeviceSynchronize();
	ExecuteFirstLayer<<<n_blocks,n_threads>>>(Layer1_Weights_GPU, Data_Layer_GPU_R, Data_Layer_GPU_G, Data_Layer_GPU_B, Layer1_Features);
	
	err = hipGetLastError();
        if (err != hipSuccess)
        {
        	fprintf(stderr, "1st LayerKernel execution failed (error code %s)\n", hipGetErrorString(err));
       		exit(EXIT_FAILURE);
        }
	hipDeviceSynchronize();
	double *Pool_Layer_Features;
	err = hipMalloc((void**) &Pool_Layer_Features, 32*16*16* NUM * sizeof(double));
	pooling1<<<n_blocks,n_threads>>>(Layer1_Features, Pool_Layer_Features, 32, 16, 16, 5, 2, 32, 32);
	hipDeviceSynchronize();
	err = hipGetLastError();
        if (err != hipSuccess)
        {
        	fprintf(stderr, "1st Pool Kernel execution failed (error code %s)\n", hipGetErrorString(err));
       		exit(EXIT_FAILURE);
        }
	double *Layer2_Weights_GPU;
	err = hipMalloc((void**) &Layer2_Weights_GPU, 5*5*32*32* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	hipMemcpy(Layer2_Weights_GPU,Layer2_Weights_CPU, sizeof(double)*5*5*32*32*NUM, hipMemcpyHostToDevice);
	double *Layer2_Features;	
	err = hipMalloc((void**) &Layer2_Features, 32*16*16* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	ExecuteSecondLayer<<<n_blocks,n_threads>>>(Layer2_Weights_GPU, Layer2_Features, Pool_Layer_Features);
	hipDeviceSynchronize();
	err = hipGetLastError();
        if (err != hipSuccess)
        {
        	fprintf(stderr, "2nd Layer Kernel execution failed (error code %s)\n", hipGetErrorString(err));
       		exit(EXIT_FAILURE);
        }
	double *Pool2_Layer_Features;
	hipMalloc((void**) &Pool2_Layer_Features, 32*8*8* NUM * sizeof(double));	
	pooling2<<<n_blocks,n_threads>>>(Layer2_Features, Pool2_Layer_Features, 32, 8, 8, 5, 2, 16, 16);
	hipDeviceSynchronize();
	err = hipGetLastError();
        if (err != hipSuccess)
        {
        	fprintf(stderr, "2nd Pool Kernel execution failed (error code %s)\n", hipGetErrorString(err));
       		exit(EXIT_FAILURE);
        }
	double *Layer3_Weights_GPU;
	err = hipMalloc((void**) &Layer3_Weights_GPU, 5*5*32*64* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	hipMemcpy(Layer3_Weights_GPU,Layer3_Weights_CPU, sizeof(double)*5*5*32*64*NUM, hipMemcpyHostToDevice);
	double *Layer3_Features;	
	err = hipMalloc((void**) &Layer3_Features, 64*8*8* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	ExecuteThirdLayer<<<n_blocks,n_threads>>>(Layer3_Weights_GPU, Layer3_Features, Pool2_Layer_Features);
	hipDeviceSynchronize();
	err = hipGetLastError();
        if (err != hipSuccess)
        {
        	fprintf(stderr, "3rd Layer Kernel execution failed (error code %s)\n", hipGetErrorString(err));
       		exit(EXIT_FAILURE);
        }
	double *Pool3_Layer_Features;
	hipMalloc((void**) &Pool3_Layer_Features, 64*4*4* NUM * sizeof(double));
	pooling3<<<n_blocks,n_threads>>>(Layer3_Features, Pool3_Layer_Features, 64, 4, 4, 5, 2, 8, 8);
	hipDeviceSynchronize();
	err = hipGetLastError();
        if (err != hipSuccess)
        {
        	fprintf(stderr, "3rd Pool Kernel execution failed (error code %s)\n", hipGetErrorString(err));
       		exit(EXIT_FAILURE);
        }
	double *Layer4_Features;
	hipMalloc((void**) &Layer4_Features, 64*sizeof(double));
	double *Layer4_Weights_GPU;
	err = hipMalloc((void**) &Layer4_Weights_GPU, 64*4*4*64* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	hipMemcpy(Layer4_Weights_GPU,Layer4_Weights_CPU, sizeof(double)*64*4*4*64*NUM, hipMemcpyHostToDevice);
	ExecuteFourthLayer<<<1,64>>>(Layer4_Weights_GPU, Layer4_Features, Pool3_Layer_Features);	
	hipDeviceSynchronize();
	err = hipGetLastError();
        if (err != hipSuccess)
        {
        	fprintf(stderr, "4th Kernel execution failed (error code %s)\n", hipGetErrorString(err));
       		exit(EXIT_FAILURE);
        }
	double *Layer5_Features;
	hipMalloc((void**) &Layer5_Features, 9*sizeof(double));
	double *Layer5_Weights_GPU;
	err = hipMalloc((void**) &Layer5_Weights_GPU, 64*9* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	hipMemcpy(Layer5_Weights_GPU,Layer5_Weights_CPU, sizeof(double)*64*9*NUM, hipMemcpyHostToDevice);
	ExecuteFifthLayer<<<1,32>>>(Layer5_Weights_GPU, Layer5_Features, Layer4_Features);	
	hipDeviceSynchronize();
	err = hipGetLastError();
        if (err != hipSuccess)
        {
        	fprintf(stderr, "5th Kernel execution failed (error code %s)\n", hipGetErrorString(err));
       		exit(EXIT_FAILURE);
        }

}


